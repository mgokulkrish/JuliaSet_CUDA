#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex{
    float r;
    float i;

    __device__ hipComplex(float a, float b) : r(a), i(b) {}

    __device__ float magnitude2(void) { return r*r + i*i;}

    __device__ hipComplex operator * (const hipComplex &a){
        return hipComplex(r*a.r - i*a.i, r*a.i + i*a.r);
    }

    __device__ hipComplex operator + (const hipComplex &a){
        return hipComplex(r + a.r, i + a.i);
    }
};

__device__
int julia(int x, int y){
    const float scale = 1.5;
    int dim = DIM;
    float jx = scale * (float)(dim/2 - x)/(dim/2);
    float jy = scale * (float)(dim/2 - y)/(dim/2);

    hipComplex z(jx, jy);
    hipComplex c(-0.8, 0.156);

    for(int i=0; i<200; i++){
       z = z*z + c; 
       if(z.magnitude2() > 1000)
           return 0;
    }


    return 1;
}

__global__
void kernel(unsigned char* ptr){
    int x = blockIdx.x;
    int y = blockIdx.y;

    int offset = x + y*gridDim.x;

    int juliaValue = julia(x, y);
    ptr[offset*4 + 0] = 255*juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;

    return;
}


int main(void){
    CPUBitmap bitmap(DIM, DIM);
    unsigned char *dev_ptr;

    HANDLE_ERROR(hipMalloc((void**)&dev_ptr, bitmap.image_size()));
    dim3 grid(DIM, DIM);

    kernel<<<grid, 1>>>(dev_ptr);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_ptr, bitmap.image_size(), hipMemcpyDeviceToHost));

    bitmap.display_and_exit();
    
    hipFree(dev_ptr);
    
}
